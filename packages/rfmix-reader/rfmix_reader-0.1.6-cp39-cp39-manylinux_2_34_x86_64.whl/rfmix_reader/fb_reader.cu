/*
 * Adapted from the `_bed_reader.h` script in the `pandas-plink` package.
 * Source: https://github.com/limix/pandas-plink/blob/main/pandas_plink/_bed_reader.h
 */

#include <stdio.h>
#include <stdint.h>
#include <hip/hip_runtime.h>

#define MIN(a,b) ((a > b) ? b : a)

__global__
void read_fb_chunk_kernel(uint8_t *buff, uint64_t nrows, uint64_t ncols,
		          uint64_t row_start, uint64_t col_start, uint64_t row_end,
			  uint64_t col_end, uint8_t *out, uint64_t *strides,
			  uint64_t row_size) {
    // Thread indices within the block
    int r = blockIdx.y * blockDim.y + threadIdx.y + row_start;
    int c = blockIdx.x * blockDim.x + threadIdx.x + col_start;

    // Check if within valid data range
    if (r < row_end && c < col_end) {
        uint64_t buff_index = r * row_size + c / 4;
	char b = buff[buff_index];
	char b0 = b & 0x55;
	char b1 = (b & 0xAA) >> 1;
	char p0 = b0 ^ b1;
	char p1 = (b0 | b1) & b0;
	p1 <<= 1;
	p0 |= p1;
	uint64_t ce = MIN(c + 4, col_end);

        for (; c < ce; ++c) {
	    out[(r - row_start) * strides[0] + (c - col_start) * strides[1]] = p0 & 3;
	    p0 >>= 2;
	}
    }
}

void read_fb_chunk(uint8_t *buff, uint64_t nrows, uint64_t ncols,
		   uint64_t row_start, uint64_t col_start, uint64_t row_end,
		   uint64_t col_end, uint8_t *out, uint64_t *strides) {
    uint64_t row_size = (ncols + 3) / 4;

    // Allocate GPU memory
    uint8_t* d_buff = nullptr;
    uint8_t* d_out = nullptr;
    uint64_t* d_strides = nullptr;
    hipError_t err = hipSuccess;

    err = hipMalloc(&d_buff, nrows * row_size * sizeof(uint8_t));
    if (err != hipSuccess) {
        fprintf(stderr, "Error allocating GPU memory for d_buff: %s\n", hipGetErrorString(err));
        return;
    }

    err = hipMalloc(&d_out, (row_end - row_start) * (col_end - col_start) * sizeof(uint8_t));
    if (err != hipSuccess) {
        fprintf(stderr, "Error allocating GPU memory for d_out: %s\n", hipGetErrorString(err));
        hipFree(d_buff);
        return;
    }

    err = hipMalloc(&d_strides, 2 * sizeof(uint64_t));
    if (err != hipSuccess) {
        fprintf(stderr, "Error allocating GPU memory for d_strides: %s\n", hipGetErrorString(err));
        hipFree(d_buff);
        hipFree(d_out);
        return;
    }

    // Copy data to GPU
    err = hipMemcpy(d_buff, buff, nrows * row_size * sizeof(uint8_t), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Error copying data to GPU for d_buff: %s\n", hipGetErrorString(err));
        hipFree(d_buff);
        hipFree(d_out);
        hipFree(d_strides);
        return;
    }

    err = hipMemcpy(d_strides, strides, 2 * sizeof(uint64_t), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Error copying data to GPU for d_strides: %s\n", hipGetErrorString(err));
        hipFree(d_buff);
        hipFree(d_out);
        hipFree(d_strides);
        return;
    }

    // Define block and grid sizes
    const dim3 threadsPerBlock(16, 16);
    const dim3 numBlocks((col_end - col_start + threadsPerBlock.x - 1) / threadsPerBlock.x,
                         (row_end - row_start + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // Launch kernel
    read_fb_chunk_kernel<<<numBlocks, threadsPerBlock>>>(d_buff, nrows, ncols, row_start, col_start, row_end, col_end, d_out, d_strides, row_size);

    // Copy results back to host
    err = hipMemcpy(out, d_out, (row_end - row_start) * (col_end - col_start) * sizeof(uint8_t), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Error copying data from GPU for out: %s\n", hipGetErrorString(err));
    }

    // Free GPU memory
    hipFree(d_buff);
    hipFree(d_out);
    hipFree(d_strides);
}
